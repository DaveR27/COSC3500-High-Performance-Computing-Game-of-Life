
#include <hip/hip_runtime.h>
#include <fstream>
#include <string>
#include <chrono>
#include <string>
#include <iostream>
#include <iomanip>



//For GPU Access
int* grid = nullptr;
int* gridGPU = nullptr;
int* tempGPU = nullptr;
int width = 0;
int height = 0;


//Error checking for cuda calls
void checkError(hipError_t e)
{
   if (e != hipSuccess)
   {
      std::cerr << "CUDA error: " << int(e) << " : " << hipGetErrorString(e) << '\n';
      abort();
   }
}

//Changes the mod operator to work like python
__host__
int modFixH(int x, int dimension) {
    if (x < 0) {
        return dimension + (x % dimension);
    } else {
        return x % dimension;
    }
}



//Changes the mod operator to work like python
__device__
int modFix(int x, int dimension) {
    if (x < 0) {
        return dimension + (x % dimension);
    } else {
        return x % dimension;
    }
}

/**
* Given the state of a cell the GoL rules apply:
* - Any live call with fewer than 2 neighbors dies = underpopulation
* - Any live cell with two or three live neighbors lives on to the next gen
* - Any live cell with more than 3 live neighbors dies = overpopulation
* - Any dead cell with exactly 3 live neighbors becomes living = reproduction
* 
* After these rules have been checked, the grid is then updated.
*
* @gridGPU: Representation of the grid that lives on the GPU memory.
* @tempGPU: Representation of the grid that lives on the GPU memory that is used for inbetween states.
* @width: The width of the grid.
* @height: The height of the grid.
*/
__global__
void evolve(int* gridGPU, int* tempGPU, int width, int height) {
    //Defineing alive and dead
    int deadValue = 0;
    int aliveValue = 1;
    
    //Gpu loop set up
    int index = blockIdx.x*blockDim.x + threadIdx.x;
    int index2 = blockIdx.y*blockDim.y + threadIdx.y;

    int stride = blockDim.x*gridDim.x;
    int stride2 = blockDim.y*gridDim.y;

    //Gpu iteration of the cells universe
    for (int a=index; a < width; a += stride){
        for(int b=index2; b < height; b += stride2) {

            //Checks for alive neighbors
            int aliveCells = 0;
            for (int k=-1; k <= 1; k++) {
                for (int l=-1; l <= 1; l++) {
                    if (!(k == 0 && l == 0)) {
                        int neigh = gridGPU[(modFix(k + a,width) * width) + modFix(l + b,height)];
                        if ((neigh == aliveValue)) {
                            aliveCells++;
                        }   
                    }
                }
            }

            //Modifies the grid depending on surrounding
            if (aliveCells < 2 || aliveCells > 3) {
                tempGPU[(modFix(a, width)*width)+modFix(b, height)] = deadValue;
                
            } else if (aliveCells == 3) {
                tempGPU[(modFix(a, width)*width)+modFix(b, height)] = aliveValue;
            } 
        }
    }
}

/**
 * Sets up the grid so that every cell of the environment starts off
 * as being dead.
 * 
 * @grid: The grid that the game of life will be played on.
 * @width: The width of the environment.
 * @height: The height of the environment.
 */
void intialGrid(int* grid, int width, int height){
    for (int i = 0; i < width; ++i) {
        for (int j = 0; j < height; ++j) {
            grid[j*width + i] = 0;   
        }
    }
}

// void debugGrid(int* grid, int width, int height) {
//     for (int i = 0; i < width; ++i) {
//         for (int j = 0; j < height; ++j) {
//             std::cout<<grid[j*width + i]<<" ";   
//         }
//         std::cout<<std::endl;
//     }
//     std::cout<<std::endl;
// }

//Runs Program
int main(int argc, char const *argv[]) {
    int generations;
    std::string visOut = "NoVis";
    const std::string visStatus = "NoVis";
    

    if (argc == 6) {
        //argv[1] is the file you want to insert
        width = atoi(argv[2]); //argv[2]
        height = atoi(argv[3]); //argv[3]
        generations = atoi(argv[4]); //argv[4]
        visOut = argv[5];
    } else {
        //argv[1] is the file you want to insert
        width = 256; //argv[2]
        height = 256; //argv[3]
        generations = 50; //argv[4]
        
    }


    //Initalize the grid for the game
    grid = new int[width*height];
    intialGrid(grid, width, height);

    //Add pattern to the grid
    std::ifstream file(argv[1]);
    std::string line;
    int centering = 0;
    bool centered = false;
    int i = 0;
    while(getline(file, line)) {
        if(centered == false){
            centering += line.size();
            centered = true;
        }
        for (uint j=0; j<line.length(); j++) {
            if (line[j] == 'O') {
                grid[(((i+(width/2)) - (centering/2)) * width) + (j+(height/2) - (centering/2))] = 1;
            }
        }
        i++;
    }
    file.close();
    std::ofstream outFile;

    if (visStatus.compare(visOut)) { //For visualizing and verification
        outFile.open("game_of_life_save.txt");
    }

    
    //Allocate GPU memory for calculations
    checkError(hipMalloc(&gridGPU, width*height*sizeof(int)));
    checkError(hipMalloc(&tempGPU, width*height*sizeof(int)));

    //Copy intial array to GPU memory locations
    checkError(hipMemcpy(gridGPU, grid, width*height*sizeof(int), hipMemcpyHostToDevice));
    checkError(hipMemcpy(tempGPU, grid, width*height*sizeof(int), hipMemcpyHostToDevice));

    // assign a 2D distribution of CUDA "threads" within each CUDA "block"
    int Threads = 256;
    int Blocks = (width+Threads-1)/Threads;

    for (int i = 0; i < generations; i++) {
        //plays gol
        evolve<<<Blocks, Threads>>>(gridGPU, tempGPU, width, height);
        
        //Sync host and device
        hipDeviceSynchronize();

        // copy from GPU to CPU
        checkError(hipMemcpy(grid, tempGPU, width*height*sizeof(int), hipMemcpyDeviceToHost));
        checkError(hipMemcpy(gridGPU, grid, width*height*sizeof(int), hipMemcpyHostToDevice));

        if (visStatus.compare(visOut)) { //For visualizing and verification
            for (int i=0; i < width; i++) {
                for (int j=0; j < height; j++) {
                    outFile << grid[(modFixH(i, width)*width)+modFixH(j, height)];
                }
                outFile << std::endl;
            }
            outFile << std::endl;
        }   
    }

    if (visStatus.compare(visOut)) {
       outFile.close(); 
    }

    hipFree(gridGPU);
    hipFree(tempGPU);
    free(grid);
    
    return 0;
}